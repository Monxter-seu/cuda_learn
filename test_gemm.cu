#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <ctime>

#define OFFSET(row, col, rowLength) (row * rowLength + col)

void randMatrix(float *a, int row, int col){
        srand((unsigned int)time(NULL));
        float min = -10.0f;
        float max = 10.0f;

        for(int i = 0; i < row * col; i++)
        {       
                a[i] = min + (float)(rand()) / RAND_MAX * (max - min);
        }

        return;
}


void compareMatrix(float *a, float *b, int row, int col){
        for(int i = 0; i < row * col; i++)
        {
                if(a[i] - b[i] > 0.5f || a[i] - b[i] < -0.5f)
                        {
                                printf("The matrix diff too much \n");
                                return;
                        }
        }
        printf("The matirx are same \n");
        return;
}

void CPUgemm(float *a, float *b, float *c, int m, int k, int n){
        for(int i = 0; i < m; i++)
        {
                for(int j = 0; j < n; j++)
                {
                        for(int q = 0; q < k; q++)
                        {
                                c[OFFSET(i , j, n)] += a[OFFSET(i, q, k)] * b[OFFSET(q, j, n)]; 
                        }
                }
        }
        return;
}

//basic version--use global memory
__global__ void globalGemm(float *a, float *b, float *c, int M, int K, int N){
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        float temp =0.0f;
        for(int k = 0; k < K; k++)
        {
                temp += a[k + y * K] * b[k * N + x];
        }

        c[y * N + x] = temp;
}

//shared memory version
__global__ void sharedGemm(float *a, float *b, float *c, int M, int K, int N, int BM,
                           int BN , int BK){
        int x = threadIdx.x + blockDim.x * blockIdx.x;
        int y = threadIdx.y + blockDim.y * blockIdx.y;

        __shared__ tempA[BM * BK];
        __shared__ tempB[BK * BN];

}

int main()
{
        int M = 16;
        int N = 8;
        int m = 128;
        int n = 64;
        int k = 128;

        const int mem_size_a = m * k * sizeof(float);
        const int mem_size_b = k * n * sizeof(float);
        const int mem_size_c = m * n * sizeof(float);

        float * host_a = (float*) malloc(mem_size_a);
        float * host_b = (float*) malloc(mem_size_b);
        float * host_c = (float*) malloc(mem_size_c);
        float * host_c_cpu = (float*) malloc(mem_size_c);

        randMatrix(host_a, m, k);
        randMatrix(host_b, k, n);

        CPUgemm(host_a, host_b, host_c_cpu, m, k, n);

        float * device_a = NULL;
        float * device_b = NULL;
        float * device_c = NULL;

        hipMalloc((void**)&device_a, mem_size_a);
        hipMalloc((void**)&device_b, mem_size_b);
        hipMalloc((void**)&device_c, mem_size_c);

        hipMemcpy(device_a, host_a, mem_size_a, hipMemcpyHostToDevice);
        hipMemcpy(device_b, host_b, mem_size_b, hipMemcpyHostToDevice);


        dim3 blockDim(N,M);
        dim3 gridDim((n + N - 1) / N, (m + M - 1) / M);
        globalGemm<<<gridDim, blockDim>>>(device_a, device_b, device_c, m, k, n);

        hipMemcpy(host_c, device_c, mem_size_c, hipMemcpyDeviceToHost);

        compareMatrix(host_c_cpu, host_c, m, n);
}
